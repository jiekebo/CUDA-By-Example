#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
	float r;
	float i;
	__device__ hipComplex( float a, float b ) : r(a), i(b) {}
	__device__ float magnitude2( void ) {
		return r * r + i * i;
	}
	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r+a.r, i+a.i);
	}
};

__device__ int julia( int x, int y ) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x)/(DIM/2);
	float jy = scale * (float)(DIM/2 - y)/(DIM/2);
	
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i = 0;
	for(i=0; i<200; i++){
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

__global__ void kernel( unsigned char *ptr ) {
	// map from threadIdx/BlockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	// now calculate the value at that position
	int juliaValue = julia( x, y );
	ptr[offset*4 + 0] = 255 * juliaValue;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

int main( void ) {
	CPUBitmap bitmap( DIM, DIM );
	unsigned char *dev_bitmap;

	HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );

	dim3 grid( DIM, DIM );
	
	kernel<<<grid,1>>>( dev_bitmap );

	HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), 
				  dev_bitmap, 
				  bitmap.image_size(), 
				  hipMemcpyDeviceToHost ) );
	bitmap.display_and_exit();

	HANDLE_ERROR( hipFree( dev_bitmap ) );
}
